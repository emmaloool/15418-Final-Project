#include "hip/hip_runtime.h"
// Copyright 2015 Google Inc. All Rights Reserved.
//
// Use of this source code is governed by a BSD-style license
// that can be found in the COPYING file in the root of the source
// tree. An additional intellectual property rights grant can be found
// in the file PATENTS. All contributing project authors may
// be found in the AUTHORS file in the root of the source tree.
// -----------------------------------------------------------------------------
//
// CUDA variant of methods for lossless encoder
//
// Authors: Emma Liu (emmaliu@andrew.cmu.edu) and Kevin Geng (khg@andrew.cmu.edu)

#include "src/dsp/dsp.h"

#if defined(WEBP_USE_CUDA)
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include "src/hipcub/hipcub.hpp"
#include "src/dsp/lossless.h"
#include "src/dsp/lossless_common.h"
#include "src/enc/vp8li_enc.h"

extern "C" {

#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
            hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

//------------------------------------------------------------------------------

// Computes sampled size of 'size' when sampling using 'sampling bits'.
__device__ __inline__
static uint32_t VP8LSubSampleSize_device(uint32_t size, uint32_t sampling_bits) {
    return (size + (1 << sampling_bits) - 1) >> sampling_bits;
}

__device__ __inline__
static void MultipliersClear(VP8LMultipliers* const m) {
  m->green_to_red_ = 0;
  m->green_to_blue_ = 0;
  m->red_to_blue_ = 0;
}

__device__ __inline__
static void ColorCodeToMultipliers(uint32_t color_code,
                                               VP8LMultipliers* const m) {
  m->green_to_red_  = (color_code >>  0) & 0xff;
  m->green_to_blue_ = (color_code >>  8) & 0xff;
  m->red_to_blue_   = (color_code >> 16) & 0xff;
}

__device__ __inline__
static uint32_t MultipliersToColorCode(
    const VP8LMultipliers* const m) {
  return 0xff000000u |
         ((uint32_t)(m->red_to_blue_) << 16) |
         ((uint32_t)(m->green_to_blue_) << 8) |
         m->green_to_red_;
}

//------------------------------------------------------------------------------

__device__ __inline__ float SlowSLog2_device(float x) {
    return x * log2(x);
}

// Compute the combined Shanon's entropy for distribution {X} and {X+Y}
__device__ float CombinedShannonEntropy_seq_device(const int X[256], const int Y[256]) {
  int i;
  double retval = 0.;
  int sumX = 0, sumXY = 0;
  for (i = 0; i < 256; ++i) {
    const int x = X[i];
    if (x != 0) {
      const int xy = x + Y[i];
      sumX += x;
      retval -= SlowSLog2_device(x);
      sumXY += xy;
      retval -= SlowSLog2_device(xy);
    } else if (Y[i] != 0) {
      sumXY += Y[i];
      retval -= SlowSLog2_device(Y[i]);
    }
  }
  retval += SlowSLog2_device(sumX) + SlowSLog2_device(sumXY);
  return (float)retval;
}

// Compute the combined Shanon's entropy for distribution {X} and {X+Y}
// Note that BLOCK_THREADS is set to the tile width; if it's larger you might need
//   to change the template parameters to BlockReduce{Int,Float}T.
#define BLOCK_THREADS 32
#define ITEMS_PER_THREAD (256/BLOCK_THREADS)
__device__ float CombinedShannonEntropy_device(const int X[256], const int Y[256]) {

    // Specialize BlockReduce for a 1D block of 256 threads on type int
    // TODO: for some reason, BLOCK_REDUCE_WARP_REDUCTIONS produces wrong result. Why?
    typedef hipcub::BlockReduce<int, BLOCK_THREADS, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduceIntT;
    typedef hipcub::BlockReduce<float, BLOCK_THREADS, hipcub::BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY> BlockReduceFloatT;

    __shared__ double entropy_result;
    __shared__ union {
        typename BlockReduceIntT::TempStorage reduce_int;
        typename BlockReduceFloatT::TempStorage reduce_float;
    } temp;

    const int index = threadIdx.y * blockDim.x + threadIdx.x;

    int my_x_data[ITEMS_PER_THREAD];
    int my_y_data[ITEMS_PER_THREAD];
    float my_entropy_data[ITEMS_PER_THREAD];

    // ==============================================================
    // Compute sumX and sumY

    if (index < BLOCK_THREADS) {
        hipcub::LoadDirectBlocked(index, X, my_x_data);
        hipcub::LoadDirectBlocked(index, Y, my_y_data);
    }

    int sumX = BlockReduceIntT(temp.reduce_int).Sum(my_x_data);
    int sumY = BlockReduceIntT(temp.reduce_int).Sum(my_y_data);
    __syncthreads();

    // ==============================================================
    // Compute entropy

    if (index < BLOCK_THREADS) {
        for (int j = 0; j < ITEMS_PER_THREAD; j++) {
            const int x = my_x_data[j];
            const int xy = x + my_y_data[j];
            float entropy = 0.;
            if (x != 0) {
                entropy -= SlowSLog2_device(x);
            }
            if (xy != 0) {
                entropy -= SlowSLog2_device(xy);
            }
            my_entropy_data[j] = entropy;
        }
    }
    __syncthreads();

    float sumEntropy = BlockReduceFloatT(temp.reduce_float).Sum(my_entropy_data);
    __syncthreads();

    if (index == 0) {
#if 0
        // NOTE: Correctness checking commented out
        int my_sumX = 0;
        int my_sumY = 0;
        for (int i = 0; i < 256; i++) { my_sumX += X[i]; my_sumY += Y[i]; }
        if (sumX != my_sumX) printf("BADDDDDDDDDd X %d != %d\n", sumX, my_sumX);
        if (sumY != my_sumY) printf("BADDDDDDDDDd Y %d != %d\n", sumY, my_sumY);
#endif
        entropy_result =
            SlowSLog2_device(sumX) +
            SlowSLog2_device(sumX + sumY) +
            sumEntropy;
    }
    __syncthreads();

    // ==============================================================
    // Return final result
    return (float) entropy_result;
}
#undef BLOCK_THREADS
#undef ITEMS_PER_THREAD

/* 
    !!! TODO #2 !!!
    Transform PredictionCostSpatial_device helper to use CUB primitives, just like CombinedShannonEntropy_device
*/
__device__ __inline__ float PredictionCostSpatial_device(
    const int counts[256], int weight_0, double exp_val) {
  const int significant_symbols = 256 >> 4;
  const double exp_decay_factor = 0.6;
  double bits = weight_0 * counts[0];
  int i;
  for (i = 1; i < significant_symbols; ++i) {
    bits += exp_val * (counts[i] + counts[256 - i]);
    exp_val *= exp_decay_factor;
  }
  return (float)(-0.1 * bits);
}

/* 
    !!! TODO #3 !!!
    Transform PredictionCostCrossColor_device to have each thread do the relevant work, just like CopyTileWithColorTransform
*/

__device__ __inline__ float PredictionCostCrossColor_device(
        const int accumulated[256], const int counts[256]) {

    // Favor low entropy, locally and globally.
    // Favor small absolute values for PredictionCostSpatial
    static const double kExpValue = 2.4;

    float result_entropy = CombinedShannonEntropy_device(counts, accumulated);

    __shared__ float result_spatial;
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        result_spatial = PredictionCostSpatial_device(counts, 3, kExpValue);
    }
    __syncthreads();

#if 0
    // NOTE: Correctness checking commented out
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        float other_result =
            CombinedShannonEntropy_seq_device(counts, accumulated);
        if (abs(result_entropy - other_result) > 1.) {
            printf("RESULT VERY DIFFERENT: %f, %f\n", result_entropy, other_result);
        }
    }
#endif

    return result_entropy + result_spatial;
}

//------------------------------------------------------------------------------
// Red functions

__device__ __inline__ int ColorTransformDelta(int8_t color_pred, int8_t color) {
    return ((int)color_pred * color) >> 5;
}

__device__ __inline__  int8_t U32ToS8(uint32_t v) {
    return (int8_t)(v & 0xff);
}

__device__ __inline__ uint8_t TransformColorRed_device(
        uint8_t green_to_red, uint32_t argb) {
    const int8_t green = U32ToS8(argb >> 8);
    int new_red = argb >> 16;
    new_red -= ColorTransformDelta(green_to_red, green);
    return (new_red & 0xff);
}

__device__ __inline__ void CollectColorRedTransforms_device(
                                 const uint32_t* argb, int stride,
                                 int tile_width, int tile_height,
                                 int green_to_red, int histo[]) {

    // Position inside block (assume tile == block)
    int x = threadIdx.x;
    int y = threadIdx.y;

    if (x < tile_width && y < tile_height) {
        int transform_index = TransformColorRed_device(
            (uint8_t)green_to_red, argb[stride * y + x]);
        atomicAdd(&histo[transform_index], 1);
    }
    __syncthreads();
}


__device__ __inline__
static float GetPredictionCostCrossColorRed_device(
        const uint32_t* argb, int stride, int tile_width, int tile_height,
        VP8LMultipliers prev_x, VP8LMultipliers prev_y, int green_to_red,
        const int accumulated_red_histo[256]) {

    const int ind = threadIdx.y * tile_width + threadIdx.x;
    __shared__ int histo[256];
    if (ind < 256) {
        histo[ind] = 0;
    }
    __syncthreads();

    // NOTE: this work has been split over threads
    CollectColorRedTransforms_device(
        argb, stride, tile_width, tile_height,
        green_to_red, histo);

    float cur_diff;

    // TODO: this work is currently duplicated by all threads
    cur_diff = PredictionCostCrossColor_device(accumulated_red_histo, histo);
    if ((uint8_t)green_to_red == prev_x.green_to_red_) {
        cur_diff -= 3;  // favor keeping the areas locally similar
    }
    if ((uint8_t)green_to_red == prev_y.green_to_red_) {
        cur_diff -= 3;  // favor keeping the areas locally similar
    }
    if (green_to_red == 0) {
        cur_diff -= 3;
    }
    __syncthreads();

    return cur_diff;
}


// Note: this function is unchanged (pass-through)
__device__ __inline__
static void GetBestGreenToRed_device(
        const uint32_t* argb, int stride, int tile_width, int tile_height,
        VP8LMultipliers prev_x, VP8LMultipliers prev_y, int quality,
        const int accumulated_red_histo[256], VP8LMultipliers* const best_tx) {

    const int kMaxIters = 4 + ((7 * quality) >> 8);  // in range [4..6]
    int green_to_red_best = 0;
    int iter, offset;
    float best_diff = GetPredictionCostCrossColorRed_device(
        argb, stride, tile_width, tile_height, prev_x, prev_y,
        green_to_red_best, accumulated_red_histo);
    for (iter = 0; iter < kMaxIters; ++iter) {
        // ColorTransformDelta is a 3.5 bit fixed point, so 32 is equal to
        // one in color computation. Having initial delta here as 1 is sufficient
        // to explore the range of (-2, 2).
        const int delta = 32 >> iter;
        // Try a negative and a positive delta from the best known value.
        for (offset = -delta; offset <= delta; offset += 2 * delta) {
            const int green_to_red_cur = offset + green_to_red_best;
            const float cur_diff = GetPredictionCostCrossColorRed_device(
                argb, stride, tile_width, tile_height, prev_x, prev_y,
                green_to_red_cur, accumulated_red_histo);
            if (cur_diff < best_diff) {
                best_diff = cur_diff;
                green_to_red_best = green_to_red_cur;
            }
        }
    }

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        best_tx->green_to_red_ = (green_to_red_best & 0xff);
    }
    __syncthreads();
}


//------------------------------------------------------------------------------
// Blue functions

__device__ __inline__ uint8_t TransformColorBlue_device(
        uint8_t green_to_blue, uint8_t red_to_blue, uint32_t argb) {
    const int8_t green = U32ToS8(argb >>  8);
    const int8_t red   = U32ToS8(argb >> 16);
    uint8_t new_blue = argb & 0xff;
    new_blue -= ColorTransformDelta(green_to_blue, green);
    new_blue -= ColorTransformDelta(red_to_blue, red);
    return (new_blue & 0xff);
}

__device__ __inline__ void CollectColorBlueTransforms_device(
        const uint32_t* argb, int stride,
        int tile_width, int tile_height,
        int green_to_blue, int red_to_blue, int histo[]) {

    // Position inside block (assume tile == block)
    int x = threadIdx.x;
    int y = threadIdx.y;

    if (x < tile_width && y < tile_height) {
        int transform_index = TransformColorBlue_device(
            (uint8_t)green_to_blue, (uint8_t)red_to_blue,
            argb[stride * y + x]);
        atomicAdd(&histo[transform_index], 1);
    }
    __syncthreads();
}

__device__
static float GetPredictionCostCrossColorBlue_device(
    const uint32_t* argb, int stride, int tile_width, int tile_height,
    VP8LMultipliers prev_x, VP8LMultipliers prev_y,
    int green_to_blue, int red_to_blue, const int accumulated_blue_histo[256]) {

    const int ind = threadIdx.y * tile_width + threadIdx.x;
    __shared__ int histo[256];
    if (ind < 256) {
        histo[ind] = 0;
    }
    __syncthreads();

    // NOTE: this work is parallelized over threads
    CollectColorBlueTransforms_device(
        argb, stride, tile_width, tile_height,
        green_to_blue, red_to_blue, histo);

    __shared__ float cur_diff;

    // TODO: this work is currently duplicated by all threads
    cur_diff = PredictionCostCrossColor_device(accumulated_blue_histo, histo);
    if ((uint8_t)green_to_blue == prev_x.green_to_blue_) {
        cur_diff -= 3;  // favor keeping the areas locally similar
    }
    if ((uint8_t)green_to_blue == prev_y.green_to_blue_) {
        cur_diff -= 3;  // favor keeping the areas locally similar
    }
    if ((uint8_t)red_to_blue == prev_x.red_to_blue_) {
        cur_diff -= 3;  // favor keeping the areas locally similar
    }
    if ((uint8_t)red_to_blue == prev_y.red_to_blue_) {
        cur_diff -= 3;  // favor keeping the areas locally similar
    }
    if (green_to_blue == 0) {
        cur_diff -= 3;
    }
    if (red_to_blue == 0) {
        cur_diff -= 3;
    }
    __syncthreads();

    return cur_diff;
}

#define kGreenRedToBlueNumAxis 8
#define kGreenRedToBlueMaxIters 7

__device__
static void GetBestGreenRedToBlue_device(
    const uint32_t* argb, int stride, int tile_width, int tile_height,
    VP8LMultipliers prev_x, VP8LMultipliers prev_y, int quality,
    const int accumulated_blue_histo[256],
    VP8LMultipliers* const best_tx) {
  const int8_t offset[kGreenRedToBlueNumAxis][2] =
      {{0, -1}, {0, 1}, {-1, 0}, {1, 0}, {-1, -1}, {-1, 1}, {1, -1}, {1, 1}};
  const int8_t delta_lut[kGreenRedToBlueMaxIters] = { 16, 16, 8, 4, 2, 2, 2 };
  const int iters =
      (quality < 25) ? 1 : (quality > 50) ? kGreenRedToBlueMaxIters : 4;
  int green_to_blue_best = 0;
  int red_to_blue_best = 0;
  int iter;
  // Initial value at origin:
  float best_diff = GetPredictionCostCrossColorBlue_device(
      argb, stride, tile_width, tile_height, prev_x, prev_y,
      green_to_blue_best, red_to_blue_best, accumulated_blue_histo);
  for (iter = 0; iter < iters; ++iter) {
    const int delta = delta_lut[iter];
    int axis;
    for (axis = 0; axis < kGreenRedToBlueNumAxis; ++axis) {
      const int green_to_blue_cur =
          offset[axis][0] * delta + green_to_blue_best;
      const int red_to_blue_cur = offset[axis][1] * delta + red_to_blue_best;
      const float cur_diff = GetPredictionCostCrossColorBlue_device(
          argb, stride, tile_width, tile_height, prev_x, prev_y,
          green_to_blue_cur, red_to_blue_cur, accumulated_blue_histo);
      if (cur_diff < best_diff) {
        best_diff = cur_diff;
        green_to_blue_best = green_to_blue_cur;
        red_to_blue_best = red_to_blue_cur;
      }
      if (quality < 25 && iter == 4) {
        // Only axis aligned diffs for lower quality.
        break;  // next iter.
      }
    }
    if (delta == 2 && green_to_blue_best == 0 && red_to_blue_best == 0) {
      // Further iterations would not help.
      break;  // out of iter-loop.
    }
  }

  if (threadIdx.x == 0 && threadIdx.y == 0) {
      best_tx->green_to_blue_ = green_to_blue_best & 0xff;
      best_tx->red_to_blue_ = red_to_blue_best & 0xff;
  }
  __syncthreads();
}
#undef kGreenRedToBlueMaxIters
#undef kGreenRedToBlueNumAxis

//------------------------------------------------------------------------------
// GetBestColorTransformForTile + Subroutines

__device__ __inline__
static VP8LMultipliers GetBestColorTransformForTile_device(
        int tile_x, int tile_y, int bits,
        VP8LMultipliers prev_x,
        VP8LMultipliers prev_y,
        int quality, int xsize, int ysize,
        const int accumulated_red_histo[256],
        const int accumulated_blue_histo[256],
        const uint32_t* const argb) {

    const int max_tile_size = 1 << bits;
    const int tile_y_offset = tile_y * max_tile_size;
    const int tile_x_offset = tile_x * max_tile_size;
    const int all_x_max = min(tile_x_offset + max_tile_size, xsize);
    const int all_y_max = min(tile_y_offset + max_tile_size, ysize);
    const int tile_width = all_x_max - tile_x_offset;
    const int tile_height = all_y_max - tile_y_offset;
    const uint32_t* const tile_argb = argb + tile_y_offset * xsize + tile_x_offset;

    __shared__ VP8LMultipliers best_tx;
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        MultipliersClear(&best_tx);
    }
    __syncthreads();

    GetBestGreenToRed_device(
            tile_argb, xsize, tile_width, tile_height,
            prev_x, prev_y, quality, accumulated_red_histo, &best_tx);

    GetBestGreenRedToBlue_device(
            tile_argb, xsize, tile_width, tile_height,
            prev_x, prev_y, quality, accumulated_blue_histo,
            &best_tx);

    return best_tx;
}

//------------------------------------------------------------------------------
// CopyTileWithColorTransform - unserialized, now each thread computes its works
// in the scope of copying when launched from ColorSpaceTransform

__device__ __inline__
static void CopyTileWithColorTransform_device(int xsize, int ysize,
                                              int tile_x, int tile_y,
                                              int max_tile_size,
                                              VP8LMultipliers color_transform,
                                              uint32_t* data) {

    // Overall index from position of thread in current block, and given the block we are in.
    int x = blockIdx.x * blockDim.x + threadIdx.x;    //equivalent to i counter
    int y = blockIdx.y * blockDim.y + threadIdx.y;    //equivalent to yscan counter

    // Calculate bounding values, offset into argb
    const int xscan = min(max_tile_size, xsize - tile_x);       // 0 <= i < xscan
    const int yscan = min(max_tile_size, ysize - tile_y);     // yscan = y > 0
    data += tile_y * xsize + tile_x;

    if (x >= xscan || y >= yscan) return;

    const uint32_t argb = data[xsize * y + x];      // Adjusted x, to account for data += xsize
    const int8_t green = U32ToS8(argb >>  8);
    const int8_t red   = U32ToS8(argb >> 16);
    int new_red = red & 0xff;
    int new_blue = argb & 0xff;
    new_red -= ColorTransformDelta(color_transform.green_to_red_, green);
    new_red &= 0xff;
    new_blue -= ColorTransformDelta(color_transform.green_to_blue_, green);
    new_blue -= ColorTransformDelta(color_transform.red_to_blue_, red);
    new_blue &= 0xff;
    data[xsize * y + x] = (argb & 0xff00ff00u) | (new_red << 16) | (new_blue);
}

//------------------------------------------------------------------------------
// ColorSpaceTransform

__global__ void
__launch_bounds__(1024)
ColorSpaceTransform_kernel(
        int width, int height, int bits, int quality,
        uint32_t* const argb, uint32_t* image,
        int accumulated_red_histo[256], int accumulated_blue_histo[256]) {

    const int tile_x = blockIdx.x;
    const int tile_y = blockIdx.y;

    const int max_tile_size = 1 << bits;
    const int tile_xsize = VP8LSubSampleSize_device(width, bits);
    const int tile_ysize = VP8LSubSampleSize_device(height, bits);

    // TODO: prev_x and prev_y are always zeroed here
    __shared__ VP8LMultipliers prev_x, prev_y;
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        MultipliersClear(&prev_y);
        MultipliersClear(&prev_x);
    }
    __syncthreads();

    const int tile_x_offset = tile_x * max_tile_size;
    const int tile_y_offset = tile_y * max_tile_size;
    const int all_x_max = min(tile_x_offset + max_tile_size, width);
    const int all_y_max = min(tile_y_offset + max_tile_size, height);
    const int offset = tile_y * tile_xsize + tile_x;
    // TODO: disabled to avoid nondeterminism
    //if (threadIdx.x == 0 && threadIdx.y == 0) {
    //    if (tile_y != 0) {
    //        ColorCodeToMultipliers(image[offset - tile_xsize], &prev_y);
    //    }
    //}
    //__syncthreads();

    // Note that device_accumulated_red_histo is passed as const.
    // So it won't be changed by this function call

    prev_x = GetBestColorTransformForTile_device(
            tile_x, tile_y, bits,
            prev_x, prev_y,
            quality, width, height,
            accumulated_red_histo,
            accumulated_blue_histo,
            argb);

    // Parallelizing CopyTileWithColorTransform_device...
    CopyTileWithColorTransform_device(
                width, height, tile_x_offset, tile_y_offset,
                max_tile_size, prev_x, argb);

    __syncthreads();

    // Gather accumulated histogram data.
    // TODO: this is disabled due to nondeterminism
    //int y = tile_y_offset + threadIdx.y;
    //int ix = y * width + tile_x_offset + threadIdx.x;
    //int ix_end = y * width + all_x_max;

    //if (y < all_y_max && ix < ix_end) {
    //    const uint32_t pix = argb[ix];
    //    bool skip =
    //        (ix >= 2 && pix == argb[ix - 2] && pix == argb[ix - 1])
    //            // repeated pixels are handled by backward references
    //        ||
    //        (ix >= width + 2 && argb[ix - 2] == argb[ix - width - 2] &&
    //            argb[ix - 1] == argb[ix - width - 1] && pix == argb[ix - width])
    //            // repeated pixels are handled by backward references
    //        ;
    //    if (!skip) {
    //        ++accumulated_red_histo[(pix >> 16) & 0xff];
    //        ++accumulated_blue_histo[(pix >> 0) & 0xff];
    //    }
    //}
    //__syncthreads();
}


void VP8LColorSpaceTransform_CUDA(int width, int height, int bits, int quality,
                               uint32_t* const argb, uint32_t* image) {

    const int max_tile_size = 1 << bits;
    const int tile_xsize = VP8LSubSampleSize(width, bits);
    const int tile_ysize = VP8LSubSampleSize(height, bits);

    assert(max_tile_size == 32);
    assert(max_tile_size == 32);

    // Allocate device_argb, copying from argb
    uint32_t *device_argb;
    cudaCheckError(hipMalloc(&device_argb, width * height * sizeof(uint32_t)));
    cudaCheckError(hipMemcpy(device_argb, argb, width * height * sizeof(uint32_t), hipMemcpyHostToDevice));

    // Allocate device_image, uninitialized
    uint32_t *device_image;
    cudaCheckError(hipMalloc(&device_image, tile_xsize * tile_ysize * sizeof(uint32_t)));

    // Allocate device_accumulated_red_histo, zeroed
    int *device_accumulated_red_histo;
    cudaCheckError(hipMalloc(&device_accumulated_red_histo, 256 * sizeof(*device_accumulated_red_histo)));
    cudaCheckError(hipMemset(device_accumulated_red_histo, 0, 256 * sizeof(*device_accumulated_red_histo)));

    // Allocate device_accumulated_blue_histo, zeroed
    int *device_accumulated_blue_histo;
    cudaCheckError(hipMalloc(&device_accumulated_blue_histo, 256 * sizeof(*device_accumulated_blue_histo)));
    cudaCheckError(hipMemset(device_accumulated_blue_histo, 0, 256 * sizeof(*device_accumulated_blue_histo)));

    // Perform kernel launch
    dim3 blockDim(max_tile_size, max_tile_size);
    dim3 gridDim(tile_xsize, tile_ysize);

    ColorSpaceTransform_kernel<<<gridDim, blockDim>>>(
        width, height, bits, quality,
        device_argb, device_image,
        device_accumulated_red_histo, device_accumulated_blue_histo);

    cudaCheckError(hipPeekAtLastError());
    cudaCheckError(hipMemcpy(argb, device_argb, width * height * sizeof(uint32_t), hipMemcpyDeviceToHost));
    cudaCheckError(hipMemcpy(image, device_image, tile_xsize * tile_ysize * sizeof(uint32_t), hipMemcpyDeviceToHost));

    cudaCheckError(hipFree(device_argb));
    cudaCheckError(hipFree(device_image));
    cudaCheckError(hipFree(device_accumulated_red_histo));
    cudaCheckError(hipFree(device_accumulated_blue_histo));
}

/*
    !!! TODO #4 !!!
    Write + run script to run cwebp on all images in the folders (+ maybe get initial reference timings for ColorTransform overall)
*/


} // extern "C"

#endif // WEBP_USE_CUDA


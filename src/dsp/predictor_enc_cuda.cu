// Copyright 2015 Google Inc. All Rights Reserved.
//
// Use of this source code is governed by a BSD-style license
// that can be found in the COPYING file in the root of the source
// tree. An additional intellectual property rights grant can be found
// in the file PATENTS. All contributing project authors may
// be found in the AUTHORS file in the root of the source tree.
// -----------------------------------------------------------------------------
//
// CUDA variant of methods for lossless encoder
//
// Authors: Emma Liu (emmaliu@andrew.cmu.edu) and Kevin Geng (khg@andrew.cmu.edu)

#include "src/dsp/dsp.h"

#if defined(WEBP_USE_CUDA)
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>
#include "src/dsp/lossless.h"
#include "src/dsp/lossless_common.h"
#include "src/enc/vp8li_enc.h"

//------------------------------------------------------------------------------

static WEBP_INLINE void MultipliersClear(VP8LMultipliers* const m) {
  m->green_to_red_ = 0;
  m->green_to_blue_ = 0;
  m->red_to_blue_ = 0;
}

static WEBP_INLINE uint32_t VP8LSubSampleSize(uint32_t size,
                                              uint32_t sampling_bits) {
  return (size + (1 << sampling_bits) - 1) >> sampling_bits;
}

static WEBP_INLINE int GetMin(int a, int b) { return (a > b) ? b : a; }

static WEBP_INLINE void ColorCodeToMultipliers(uint32_t color_code,
                                               VP8LMultipliers* const m) {
  m->green_to_red_  = (color_code >>  0) & 0xff;
  m->green_to_blue_ = (color_code >>  8) & 0xff;
  m->red_to_blue_   = (color_code >> 16) & 0xff;
}

static WEBP_INLINE uint32_t MultipliersToColorCode(
    const VP8LMultipliers* const m) {
  return 0xff000000u |
         ((uint32_t)(m->red_to_blue_) << 16) |
         ((uint32_t)(m->green_to_blue_) << 8) |
         m->green_to_red_;
}


//------------------------------------------------------------------------------

static float PredictionCostSpatial(const int counts[256], int weight_0,
                                   double exp_val) {
  const int significant_symbols = 256 >> 4;
  const double exp_decay_factor = 0.6;
  double bits = weight_0 * counts[0];
  int i;
  for (i = 1; i < significant_symbols; ++i) {
    bits += exp_val * (counts[i] + counts[256 - i]);
    exp_val *= exp_decay_factor;
  }
  return (float)(-0.1 * bits);
}

static float PredictionCostCrossColor(const int accumulated[256],
                                      const int counts[256]) {
  // Favor low entropy, locally and globally.
  // Favor small absolute values for PredictionCostSpatial
  static const double kExpValue = 2.4;
  return VP8LCombinedShannonEntropy(counts, accumulated) +
         PredictionCostSpatial(counts, 3, kExpValue);
}

//------------------------------------------------------------------------------
// Red functions

static float GetPredictionCostCrossColorRed(
    const uint32_t* argb, int stride, int tile_width, int tile_height,
    VP8LMultipliers prev_x, VP8LMultipliers prev_y, int green_to_red,
    const int accumulated_red_histo[256]) {
  int histo[256] = { 0 };
  float cur_diff;

  VP8LCollectColorRedTransforms(argb, stride, tile_width, tile_height,
                                green_to_red, histo);

  cur_diff = PredictionCostCrossColor(accumulated_red_histo, histo);
  if ((uint8_t)green_to_red == prev_x.green_to_red_) {
    cur_diff -= 3;  // favor keeping the areas locally similar
  }
  if ((uint8_t)green_to_red == prev_y.green_to_red_) {
    cur_diff -= 3;  // favor keeping the areas locally similar
  }
  if (green_to_red == 0) {
    cur_diff -= 3;
  }
  return cur_diff;
}

static void GetBestGreenToRed(
        const uint32_t* argb, int stride, int tile_width, int tile_height,
        VP8LMultipliers prev_x, VP8LMultipliers prev_y, int quality,
        const int accumulated_red_histo[256], VP8LMultipliers* const best_tx) {
    const int kMaxIters = 4 + ((7 * quality) >> 8);  // in range [4..6]
    int green_to_red_best = 0;
    int iter, offset;
    float best_diff = GetPredictionCostCrossColorRed(
                          argb, stride, tile_width, tile_height, prev_x, prev_y,
                          green_to_red_best, accumulated_red_histo);
    for (iter = 0; iter < kMaxIters; ++iter) {
        // ColorTransformDelta is a 3.5 bit fixed point, so 32 is equal to
        // one in color computation. Having initial delta here as 1 is sufficient
        // to explore the range of (-2, 2).
        const int delta = 32 >> iter;
        // Try a negative and a positive delta from the best known value.
        for (offset = -delta; offset <= delta; offset += 2 * delta) {
            const int green_to_red_cur = offset + green_to_red_best;
            const float cur_diff = GetPredictionCostCrossColorRed(
            argb, stride, tile_width, tile_height, prev_x, prev_y,
            green_to_red_cur, accumulated_red_histo);
            if (cur_diff < best_diff) {
                best_diff = cur_diff;
                green_to_red_best = green_to_red_cur;
            }
        }
    }
    best_tx->green_to_red_ = (green_to_red_best & 0xff);
}


//------------------------------------------------------------------------------
// Blue functions

static float GetPredictionCostCrossColorBlue(
    const uint32_t* argb, int stride, int tile_width, int tile_height,
    VP8LMultipliers prev_x, VP8LMultipliers prev_y,
    int green_to_blue, int red_to_blue, const int accumulated_blue_histo[256]) {
  int histo[256] = { 0 };
  float cur_diff;

  VP8LCollectColorBlueTransforms(argb, stride, tile_width, tile_height,
                                 green_to_blue, red_to_blue, histo);

  cur_diff = PredictionCostCrossColor(accumulated_blue_histo, histo);
  if ((uint8_t)green_to_blue == prev_x.green_to_blue_) {
    cur_diff -= 3;  // favor keeping the areas locally similar
  }
  if ((uint8_t)green_to_blue == prev_y.green_to_blue_) {
    cur_diff -= 3;  // favor keeping the areas locally similar
  }
  if ((uint8_t)red_to_blue == prev_x.red_to_blue_) {
    cur_diff -= 3;  // favor keeping the areas locally similar
  }
  if ((uint8_t)red_to_blue == prev_y.red_to_blue_) {
    cur_diff -= 3;  // favor keeping the areas locally similar
  }
  if (green_to_blue == 0) {
    cur_diff -= 3;
  }
  if (red_to_blue == 0) {
    cur_diff -= 3;
  }
  return cur_diff;
}

#define kGreenRedToBlueNumAxis 8
#define kGreenRedToBlueMaxIters 7
static void GetBestGreenRedToBlue(
    const uint32_t* argb, int stride, int tile_width, int tile_height,
    VP8LMultipliers prev_x, VP8LMultipliers prev_y, int quality,
    const int accumulated_blue_histo[256],
    VP8LMultipliers* const best_tx) {
  const int8_t offset[kGreenRedToBlueNumAxis][2] =
      {{0, -1}, {0, 1}, {-1, 0}, {1, 0}, {-1, -1}, {-1, 1}, {1, -1}, {1, 1}};
  const int8_t delta_lut[kGreenRedToBlueMaxIters] = { 16, 16, 8, 4, 2, 2, 2 };
  const int iters =
      (quality < 25) ? 1 : (quality > 50) ? kGreenRedToBlueMaxIters : 4;
  int green_to_blue_best = 0;
  int red_to_blue_best = 0;
  int iter;
  // Initial value at origin:
  float best_diff = GetPredictionCostCrossColorBlue(
      argb, stride, tile_width, tile_height, prev_x, prev_y,
      green_to_blue_best, red_to_blue_best, accumulated_blue_histo);
  for (iter = 0; iter < iters; ++iter) {
    const int delta = delta_lut[iter];
    int axis;
    for (axis = 0; axis < kGreenRedToBlueNumAxis; ++axis) {
      const int green_to_blue_cur =
          offset[axis][0] * delta + green_to_blue_best;
      const int red_to_blue_cur = offset[axis][1] * delta + red_to_blue_best;
      const float cur_diff = GetPredictionCostCrossColorBlue(
          argb, stride, tile_width, tile_height, prev_x, prev_y,
          green_to_blue_cur, red_to_blue_cur, accumulated_blue_histo);
      if (cur_diff < best_diff) {
        best_diff = cur_diff;
        green_to_blue_best = green_to_blue_cur;
        red_to_blue_best = red_to_blue_cur;
      }
      if (quality < 25 && iter == 4) {
        // Only axis aligned diffs for lower quality.
        break;  // next iter.
      }
    }
    if (delta == 2 && green_to_blue_best == 0 && red_to_blue_best == 0) {
      // Further iterations would not help.
      break;  // out of iter-loop.
    }
  }
  best_tx->green_to_blue_ = green_to_blue_best & 0xff;
  best_tx->red_to_blue_ = red_to_blue_best & 0xff;
}
#undef kGreenRedToBlueMaxIters
#undef kGreenRedToBlueNumAxis

//------------------------------------------------------------------------------
// GetBestColorTransformForTile + Subroutines

static VP8LMultipliers GetBestColorTransformForTile(int tile_x, int tile_y, int bits,
                                                    VP8LMultipliers prev_x,
                                                    VP8LMultipliers prev_y,
                                                    int quality, int xsize, int ysize,
                                                    const int accumulated_red_histo[256],
                                                    const int accumulated_blue_histo[256],
                                                    const uint32_t* const argb) {

    const int max_tile_size = 1 << bits;
    const int tile_y_offset = tile_y * max_tile_size;
    const int tile_x_offset = tile_x * max_tile_size;
    const int all_x_max = GetMin(tile_x_offset + max_tile_size, xsize);
    const int all_y_max = GetMin(tile_y_offset + max_tile_size, ysize);
    const int tile_width = all_x_max - tile_x_offset;
    const int tile_height = all_y_max - tile_y_offset;
    const uint32_t* const tile_argb = argb + tile_y_offset * xsize + tile_x_offset;
    VP8LMultipliers best_tx;
    MultipliersClear(&best_tx);

    GetBestGreenToRed(tile_argb, xsize, tile_width, tile_height,
                    prev_x, prev_y, quality, accumulated_red_histo, &best_tx);
    GetBestGreenRedToBlue(tile_argb, xsize, tile_width, tile_height,
                        prev_x, prev_y, quality, accumulated_blue_histo,
                        &best_tx);
    return best_tx;
}

//------------------------------------------------------------------------------

static void CopyTileWithColorTransform(int xsize, int ysize,
                                       int tile_x, int tile_y,
                                       int max_tile_size,
                                       VP8LMultipliers color_transform,
                                       uint32_t* argb) {
  const int xscan = GetMin(max_tile_size, xsize - tile_x);
  int yscan = GetMin(max_tile_size, ysize - tile_y);
  argb += tile_y * xsize + tile_x;
  while (yscan-- > 0) {
    VP8LTransformColor(&color_transform, argb, xscan);
    argb += xsize;
  }
}

//------------------------------------------------------------------------------
// ColorSpaceTransform

void VP8LColorSpaceTransform_C(int width, int height, int bits, int quality,
                               uint32_t* const argb, uint32_t* image) {
  const int max_tile_size = 1 << bits;
  const int tile_xsize = VP8LSubSampleSize(width, bits);
  const int tile_ysize = VP8LSubSampleSize(height, bits);
  int accumulated_red_histo[256] = { 0 };
  int accumulated_blue_histo[256] = { 0 };
  int tile_x, tile_y;
  VP8LMultipliers prev_x, prev_y;
  MultipliersClear(&prev_y);
  MultipliersClear(&prev_x);
  for (tile_y = 0; tile_y < tile_ysize; ++tile_y) {
    for (tile_x = 0; tile_x < tile_xsize; ++tile_x) {
      int y;
      const int tile_x_offset = tile_x * max_tile_size;
      const int tile_y_offset = tile_y * max_tile_size;
      const int all_x_max = GetMin(tile_x_offset + max_tile_size, width);
      const int all_y_max = GetMin(tile_y_offset + max_tile_size, height);
      const int offset = tile_y * tile_xsize + tile_x;
      if (tile_y != 0) {
        ColorCodeToMultipliers(image[offset - tile_xsize], &prev_y);
      }
      prev_x = GetBestColorTransformForTile(tile_x, tile_y, bits,
                                            prev_x, prev_y,
                                            quality, width, height,
                                            accumulated_red_histo,
                                            accumulated_blue_histo,
                                            argb);
      image[offset] = MultipliersToColorCode(&prev_x);
      CopyTileWithColorTransform(width, height, tile_x_offset, tile_y_offset,
                                 max_tile_size, prev_x, argb);

      // Gather accumulated histogram data.
      for (y = tile_y_offset; y < all_y_max; ++y) {
        int ix = y * width + tile_x_offset;
        const int ix_end = ix + all_x_max - tile_x_offset;
        for (; ix < ix_end; ++ix) {
          const uint32_t pix = argb[ix];
          if (ix >= 2 &&
              pix == argb[ix - 2] &&
              pix == argb[ix - 1]) {
            continue;  // repeated pixels are handled by backward references
          }
          if (ix >= width + 2 &&
              argb[ix - 2] == argb[ix - width - 2] &&
              argb[ix - 1] == argb[ix - width - 1] &&
              pix == argb[ix - width]) {
            continue;  // repeated pixels are handled by backward references
          }
          ++accumulated_red_histo[(pix >> 16) & 0xff];
          ++accumulated_blue_histo[(pix >> 0) & 0xff];
        }
      }
    }
  }
}


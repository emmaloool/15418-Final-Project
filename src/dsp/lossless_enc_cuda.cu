// Copyright 2015 Google Inc. All Rights Reserved.
//
// Use of this source code is governed by a BSD-style license
// that can be found in the COPYING file in the root of the source
// tree. An additional intellectual property rights grant can be found
// in the file PATENTS. All contributing project authors may
// be found in the AUTHORS file in the root of the source tree.
// -----------------------------------------------------------------------------
//
// SSE2 variant of methods for lossless encoder
//
// Author: Skal (pascal.massimino@gmail.com)

#include "src/dsp/dsp.h"

#if defined(WEBP_USE_CUDA)
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "src/dsp/lossless.h"
#include "src/dsp/lossless_common.h"

//------------------------------------------------------------------------------
// Subtract-Green Transform

static void SubtractGreenFromBlueAndRed_CUDA(uint32_t* argb_data,
                                             int num_pixels) {

  printf("HELLO WORLD WE ARE CALLING CUDA FUNCTION\n");
  int i = 0;
  // fallthrough and finish off with plain-C
  if (i != num_pixels) {
    VP8LSubtractGreenFromBlueAndRed_C(argb_data + i, num_pixels - i);
  }
}

//------------------------------------------------------------------------------
// Entry point

extern "C" void VP8LEncDspInitCUDA(void);

WEBP_TSAN_IGNORE_FUNCTION void VP8LEncDspInitCUDA(void) {
  VP8LSubtractGreenFromBlueAndRed = SubtractGreenFromBlueAndRed_CUDA;
}

#else  // !WEBP_USE_SSE2

WEBP_DSP_INIT_STUB(VP8LEncDspInitCUDA)

#endif  // WEBP_USE_SSE2
